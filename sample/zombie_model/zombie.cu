#include "hip/hip_runtime.h"
#include "zombie.cuh"
#include <math.h>
#include <cutil.h>
#include "../../agent/Turtle.cuh"
#include "seed.cuh"
#include <fstream>
#include <stdio.h>


zombie::zombie(){
}

zombie::zombie(int N, int minX, int maxX, int minY, int maxY, int* field){
  srand(SEED);
  add_zombie(N, minX, maxX, minY, maxY, field);
}

zombie::~zombie(){
}

void zombie::add_zombie(int N, int minX, int maxX, int minY, int maxY, int* field){
  for (int i=0; i<N; ++i){
    int x = rand() % (maxX-minX+1);
    int y = rand() % (maxX-minX+1);
    add_turtle(x,y);
    ++field[x+((maxX-minX+1)*y)];
  }
}

void zombie::add_zombie(int x, int y, int minX, int maxX, int minY, int maxY, int *field){
  add_turtle(x,y);
  ++field[x+((maxX-minX+1)*y)];
}

void zombie::step(int N, int minX, int maxX, int minY, int maxY, int* field){
  int field_count = (maxY-minY+1)*(maxX-minX+1);
  int* x = (int*)malloc(sizeof(int)*N);
  int* y = (int*)malloc(sizeof(int)*N);
  x = get_pointer_x(0);
  y = get_pointer_y(0);
  int* devX;
  int* devY;
  int* dev_field;
  dim3 grids((N+1023)/1024, 1, 1);
  dim3 blocks(1024, 1, 1);

  /*remove before move*/
  CUDA_SAFE_CALL(hipMalloc((void**)&devX, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devY, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_field, sizeof(int)*field_count));
  CUDA_SAFE_CALL(hipMemcpy(devX, x, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devY, y, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dev_field, field, sizeof(int)*field_count, hipMemcpyHostToDevice));
  update_field_before_move<<<grids, blocks>>>(devX, devY, dev_field, N, minX, maxX, minY, maxY);
  CUDA_SAFE_CALL(hipMemcpy(field, dev_field, sizeof(int)*field_count, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(devX));
  CUDA_SAFE_CALL(hipFree(devY));
  CUDA_SAFE_CALL(hipFree(dev_field));

  /*move*/
  CUDA_SAFE_CALL(hipMalloc((void**)&devX, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devY, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMemcpy(devX, x, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devY, y, sizeof(int)*N, hipMemcpyHostToDevice));
  move<<<grids,blocks>>>(1,devX,devY,N,minX,maxX,minY,maxY);
  CUDA_SAFE_CALL(hipMemcpy(x, devX, sizeof(int)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(y, devY, sizeof(int)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(devX));
  CUDA_SAFE_CALL(hipFree(devY));

  /*add after move*/
  CUDA_SAFE_CALL(hipMalloc((void**)&devX, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devY, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_field, sizeof(int)*field_count));
  CUDA_SAFE_CALL(hipMemcpy(devX, x, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devY, y, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dev_field, field, sizeof(int)*field_count, hipMemcpyHostToDevice));
  update_field_after_move<<<grids, blocks>>>(devX, devY, dev_field, N, minX, maxX, minY, maxY);
  CUDA_SAFE_CALL(hipMemcpy(field, dev_field, sizeof(int)*field_count, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(devX));
  CUDA_SAFE_CALL(hipFree(devY));
  CUDA_SAFE_CALL(hipFree(dev_field));

  /*Add zombies whom people become in this turn.*/
  /*
  add_zombie(death_counter, minX, maxX, minY, maxY, field);
  N += death_counter;
  */
}

__global__ void update_field_before_move(int* x, int* y, int* field, int N, int minX, int maxX, int minY, int maxY){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < N){
    if (field[(y[tid]*(maxX-minX+1))+x[tid]] > 0){
      atomicSub(&field[(y[tid]*(maxX-minX+1))+x[tid]], 1);
    }
  }
}


__global__ void update_field_after_move(int* x, int* y, int* field, int N, int minX, int maxX, int minY, int maxY){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < N){
    atomicAdd(&field[(y[tid]*(maxX-minX+1))+x[tid]], 1);
  }
}


void zombie::output_zombie_info(int N){
  ofstream ofs("info_zombie.csv");
  ofs << "id,x,y" << endl;
  for(int i=0; i<N; ++i){
    ofs << i << "," << getX(i) << "," << getY(i) << endl;
  }
}
