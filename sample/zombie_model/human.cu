#include "hip/hip_runtime.h"
#include "human.cuh"
#include <cutil.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include "../../math/random.cuh"
#include "../../observer/observer.cuh"
#include "../../agent/Turtle.cuh"
#include "seed.cuh"

/*
  エージェント個人が持つIDは配列引数の番号とする。
*/

using namespace std;

__constant__ int D_SEED = SEED;

/*
  エージェント情報を初期化する
*/
human::human(int N, int minX, int maxX, int minY, int maxY){//Nはエージェント数
  int* dev_agentX = NULL;
  int* dev_agentY = NULL;
  int* dev_agentInfect = NULL;
  int* dev_agentInfectionTime = NULL;
  add_human(N);
  int *x = get_pointer_x(0);
  int *y = get_pointer_y(0);
  int *infect_P = get_pointer_infect(0);
  int *infection_time_P = get_pointer_infection_time(0);
  //infect  = NULL;
  //die = NULL;

  CUDA_SAFE_CALL(hipMalloc((void**)&dev_agentX, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_agentY, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_agentInfect, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_agentInfectionTime, sizeof(int)*N));
	
  dim3 blocks(1024, 1, 1);
  dim3 grids((N+1023)/1024, 1, 1);

  initiate_agent<<<grids, blocks>>>(dev_agentX, dev_agentY, dev_agentInfect, dev_agentInfectionTime, N, minX, maxX, minY, maxY);
	
  CUDA_SAFE_CALL(hipMemcpy(x, dev_agentX, sizeof(int)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(y, dev_agentY, sizeof(int)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(infect_P, dev_agentInfect, sizeof(int)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(infection_time_P, dev_agentInfectionTime, sizeof(int)*N, hipMemcpyDeviceToHost));

  CUDA_SAFE_CALL(hipFree(dev_agentX));
  CUDA_SAFE_CALL(hipFree(dev_agentY));
  CUDA_SAFE_CALL(hipFree(dev_agentInfect));
  CUDA_SAFE_CALL(hipFree(dev_agentInfectionTime));
}


human::~human(){
}

/*
int human::getX(int id){
  return x[id];
}

int human::getY(int id){
  return y[id];
}
*/

int human::getInfect(int id){
  return infect[id];
}

int human::getInfectionTime(int id){
  return infection_time[id];
}

/*
int* human::get_pointer_x(){
  return x;
}

int* human::get_pointer_y(){
  return y;
}
*/

int* human::get_pointer_infect(int id){
  return &infect[id];
}

int* human::get_pointer_infection_time(int id){
  return &infection_time[id];
}

void human::add_human(int N){
  add_turtle(N);
  for (int i=0; i<N; ++i){
    infect.push_back(0);
    infection_time.push_back(0);
  }
}
/*reserveはイテレータが取得できないので不可*/

void human::die(int id){
  remove_turtle(id);
  if ((infect.empty() == false) && (infection_time.empty() == false)){
    int search = 0;
    vector<int>::iterator it_infect = infect.begin();
    vector<int>::iterator it_infection_time = infection_time.begin();
    while (search != id){
      ++it_infect;
      ++it_infection_time;
      ++search;
    }
    infect.erase(it_infect);
    infection_time.erase(it_infection_time);
  }
}

__global__ void agent_infect(int infect_distance, int* world, int* devX, int* devY, int* infect, int N, int minX, int maxX, int minY, int maxY){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  //const int infect_parameter = 80;
  if (tid < N){
    if(infect[tid] != 1){
      if (world[devX[tid]+((maxX-minX+1)*devY[tid])] >= 1){
	infect[tid] = 1;
      }

      /*以下は範囲外アクセスを考慮した周囲のチェック*/
      /*
      int pos = devX[tid] + ((maxX-minX+1)*devY[tid]);
      int count = check_with_distance(world, infect_distance, pos, minX, maxX, minY, maxY);
      */
      /*周囲チェックの処理はここまで*/
      /*
      if (count > 0){
	int parameter = (Rand() % 101) + (10*count);
	if (parameter > infect_parameter){//感染条件は適当にいじってよし
	  infect[tid] = 1;
	}
      }
      else{
	int parameter = Rand() % 101;
	if (parameter > infect_parameter){
	  infect[tid] = 1;
	}
      }
      */
    }
  }
}


__global__ void update_field_before_move(int* world, int* infect, int* x, int* y, int N, int minX, int maxX){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
  if (tid < N){
    if (infect[tid] == 1){
      if (world[(y[tid]*(maxX-minX+1))+x[tid]] > 0){
	atomicSub(&world[(y[tid]*(maxX-minX+1))+x[tid]], 1);
      }
    }
  }
}

__global__ void update_field_after_move(int* world, int* infect, int* x, int* y, int N, int minX, int maxX){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
  if (tid < N){
    if (infect[tid] == 1){
      atomicAdd(&world[(y[tid]*(maxX-minX+1))+x[tid]], 1);
    }
  }
}

__global__ void update_infection_time(int* dev_infect, int* dev_infection_time, int N){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < N){
    if (dev_infect[tid] == 1){
      ++dev_infection_time[tid];
    }
  }
}


void human::step(zombie& zombies, int* field, int minX, int maxX, int minY, int maxY, int& N, int& zombieN){
  int field_count = (maxX-minX+1)*(maxY-minY+1);
  int *tmp_field = (int*)malloc(sizeof(int)*field_count);//before info
  dim3 grids((N+1023)/1024,1,1);
  dim3 blocks(1024,1,1);
  int *devX, *devY;
  int *dev_infect, *dev_infection_time;
  int *dev_field_before;//, *dev_field_after;
  int *x = get_pointer_x(0);
  int *y = get_pointer_y(0);
  int *infect = get_pointer_infect(0);
  int *infection_time = get_pointer_infection_time(0);
  

  for (int i=(int)minY;i<(int)maxY;++i){
    tmp_field[i] = field[i];
  }

  /*update infection time*/
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_infection_time, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_infect, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMemcpy(dev_infect, infect, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dev_infection_time, infection_time, sizeof(int)*N, hipMemcpyHostToDevice));
  update_infection_time<<<grids, blocks>>>(dev_infect, dev_infection_time, N);
  CUDA_SAFE_CALL(hipMemcpy(infection_time, dev_infection_time, sizeof(int)*N, hipMemcpyDeviceToHost));
  
  CUDA_SAFE_CALL(hipFree(dev_infection_time));
  CUDA_SAFE_CALL(hipFree(dev_infect));

  /*update field before move*/
  /*
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_field_after, sizeof(int)*field_count));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_infect, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devX, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devY, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMemcpy(dev_field_after, field, sizeof(int)*field_count, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dev_infect, infect, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devX, x, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devY, y, sizeof(int)*N, hipMemcpyHostToDevice));
  update_field_before_move<<<grids, blocks>>>(dev_field_after, dev_infect, devX, devY, N, minX, maxX);
  CUDA_SAFE_CALL(hipMemcpy(field, dev_field_after, sizeof(int)*field_count, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(devX));
  CUDA_SAFE_CALL(hipFree(devY));
  CUDA_SAFE_CALL(hipFree(dev_field_after));
  CUDA_SAFE_CALL(hipFree(dev_infect));
  */
  
  /*move*/
  CUDA_SAFE_CALL(hipMalloc((void**)&devX, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devY, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMemcpy(devX, x, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devY, y, sizeof(int)*N, hipMemcpyHostToDevice));
  move<<<grids,blocks>>>(2,devX,devY,N,minX,maxX,minY,maxY);
  CUDA_SAFE_CALL(hipMemcpy(x, devX, sizeof(int)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(y, devY, sizeof(int)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(devX));
  CUDA_SAFE_CALL(hipFree(devY));
  
  /*update field after move*/
  /*
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_field_after, sizeof(int)*field_count));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_infect, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devX, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devY, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMemcpy(dev_field_after, field, sizeof(int)*field_count, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dev_infect, infect, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devX, x, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devY, y, sizeof(int)*N, hipMemcpyHostToDevice));
  update_field_after_move<<<grids, blocks>>>(dev_field_after, dev_infect, devX, devY, N, minX, maxX);
  CUDA_SAFE_CALL(hipMemcpy(field, dev_field_after, sizeof(int)*field_count, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(devX));
  CUDA_SAFE_CALL(hipFree(devY));
  CUDA_SAFE_CALL(hipFree(dev_field_after));
  CUDA_SAFE_CALL(hipFree(dev_infect));
  */

  /*infect*/
  CUDA_SAFE_CALL(hipMalloc((void**)&devX, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&devY, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMemcpy(devX, x, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(devY, y, sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_infect, sizeof(int)*N));
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_field_before, sizeof(int)*field_count));
  CUDA_SAFE_CALL(hipMemcpy(dev_infect, &infect[0], sizeof(int)*N, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dev_field_before, tmp_field, sizeof(int)*field_count, hipMemcpyHostToDevice));
  agent_infect<<<grids, blocks>>>(1, dev_field_before, devX, devY, dev_infect, N, minX, maxX, minY, maxY);
  CUDA_SAFE_CALL(hipMemcpy(infect, dev_infect, sizeof(int)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(tmp_field, dev_field_before, sizeof(int)*field_count, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(dev_infect));
  CUDA_SAFE_CALL(hipFree(dev_field_before));
  CUDA_SAFE_CALL(hipFree(devX));
  CUDA_SAFE_CALL(hipFree(devY));
  
  free(tmp_field);
  for (int id=0; id<N; ++id){
    if (infection_time[id] >= 5){
      zombies.add_zombie(getX(id), getY(id), minX, maxX, minY, maxY, field);
      //cout << "X:" << getX(id) << ", Y:" << getY(id) << endl;
      ++zombieN;
      die(id);
      --N;
      --id;
      //id = 0;
    }
  }
}



__global__ void initiate_agent(int* x, int* y, int* infect, int* infection_time, int N, int minX, int maxX, int minY, int maxY){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
	
  if (tid < N){
    //x[tid] = (Rand() << tid) % (maxX-minX+1);//% (maxX+1);
    //y[tid] = (Rand() >> tid) % (maxY-minY+1);//% (maxY+1);
    x[tid] = abs(Xorshift128(tid+D_SEED,N-tid+D_SEED) % (maxX-minX+1));
    y[tid] = abs(Xorshift128(N-tid+D_SEED,tid+D_SEED) % (maxY-minY+1));
    infect[tid] = 0;
    infection_time[tid] = 0;
  }
}

void human::output_human_info(int N){
  int infect_count = 0;
  ofstream ofs("info_human.csv");
  ofs << "id,x,y,infected,infection time" << endl;
  for (int i=0; i<N; ++i){
    if (getInfect(i) == 1){
      ++infect_count;
      ofs << i << "," << getX(i) << "," << getY(i) << "," << "true" << "," << infection_time[i] << endl;//<< getInfectionTime(i) << endl;
    }
    else{
      ofs << i << "," << getX(i) << "," << getY(i) << "," << "false" << "," << getInfectionTime(i) << endl;
    }
    
  }
  ofs << endl;
  ofs << "感染者合計：," << infect_count << "人" << endl;
}
