#include <stdio.h>
#include "human.cuh"
#include "zombie.cuh"
#include <cutil.h>
#include "../../world/worldCreator.cuh"
#include "../../agent/output.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>

using namespace std;

void run(void);

void input_agent_data(int& humanN, int& zombieN);

int main(void){
  CUDA_SAFE_CALL(hipSetDevice(0));
  float elapsed_time_ms = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  run();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  printf("Run time from start to end : %f [sec]\n", elapsed_time_ms/1000);
  
  return 0;
}

void run(void){
  worldCreator<int> world;
  int maxX = world.getMax_X();
  int minX = world.getMin_X();
  int maxY = world.getMax_Y();
  int minY = world.getMin_Y();
  //cout << "maxX:" << maxX << ", minX:" << minX << ", maxY:" << maxY << ", minY:" << minY << endl;
  world.make_2D_World();
  int* field = world.getField();
  int humanN;// = 10;
  int zombieN;// = 5;
  input_agent_data(humanN, zombieN);
  human agent(humanN, minX, maxX, minY, maxY);
  zombie zombies(zombieN, minX, maxX, minY, maxY, field);

  printf("maxX:%d, minX:%d, maxY:%d, minY:%d \n", maxX,minX,maxY,minY);//check
  printf("human=%d, zombie=%d\n", humanN, zombieN);//check
  cout << "--------------------------" << endl;
  cout << "Simulation starts!" << endl;
  /*
  cout << "initial" << endl;
  for (int i=0; i<humanN; ++i){
    cout << "x[" << i << "]=" << agent.getX(i) << ", ";
    cout << "y[" << i << "]=" << agent.getY(i) << endl;
  }
  */
  //make_output_coordinates(agent.get_pointer_x(0), agent.get_pointer_y(0), N);
  const int step_count = 100;
  for(int i=0; i<step_count; ++i){
    cout << "step " << (i+1) << endl;
    //int death_counter = 0;
    agent.step(zombies, field, minX, maxX, minY, maxY, humanN, zombieN);
    zombies.step(zombieN, minX, maxX, minY, maxY, field);
  }
  cout << "fin" << endl;
  cout << "------------------------------" << endl;
  cout << "<result>" << endl;
  printf("maxX:%d, minX:%d, maxY:%d, minY:%d \n", maxX,minX,maxY,minY);
  cout << "the sum of agents=" << (humanN+zombieN) << endl;
  cout << "simulation steps=" << step_count << endl;
  cout << "left human=" << humanN << endl;
  cout << "zombie=" << zombieN << endl;

  /*
  for (int i=0; i<humanN; ++i){
    cout << "x[" << i << "]=" << agent.getX(i) << ", ";
    cout << "y[" << i << "]=" << agent.getY(i) << endl;
  }
  */
  //output_agent_info(agent.get_pointer_x(0), agent.get_pointer_y(0), agent.get_pointer_infect(0), N);
  agent.output_human_info(humanN);
  zombies.output_zombie_info(zombieN);
  cout << "Simulation end!" << endl;
}


void input_agent_data(int& humanN, int& zombieN){
  string str, human, zombie;
  ifstream ifs("agent.props");
  getline(ifs, str);
  getline(ifs, str);
  human = str;
  getline(ifs, str);
  getline(ifs, str);
  zombie = str;

  /*文字列データを数値へ変換*/
  stringstream ss;
  ss << human;
  ss >> humanN;
  ss.clear();
  ss.str("");
  ss << zombie;
  ss >> zombieN;
  ss.clear();
  ss.str("");
}
