#include "hip/hip_runtime.h"
#include "cell.cuh"
#include <cutil.h>
#include "../../math/random.cuh"
#include <stdio.h>
#include "../../gpu/gpu.cuh"
#include "../../observer/observer.cuh"
#include <cstdlib>
#include <ctime>


using namespace std;

Cell::Cell(worldCreator<int> &world){
  this->maxX = world.getMax_X();
  this->minX = world.getMin_X();
  this->maxY = world.getMax_Y();
  this->minY = world.getMin_Y();
  int agent_size = (maxX-minX+1)*(maxY-minY+1);
  //srand(time(NULL));
  srand(1);
  for (int i=0; i<agent_size; ++i){
    this->condition.push_back((rand()%2));
  }
  world.setField(&(this->condition[0]), agent_size);
}

Cell::~Cell(){
}

int Cell::get_cell_condition(int id){
  return this->condition[id];
}

int* Cell::get_cell_condition_pointer(int id){
  return &(this->condition[id]);
}

void Cell::step(int agent_size, worldCreator<int> &world){
  dim3 grids((agent_size+1023)/1024, 1, 1);
  dim3 blocks(1024, 1, 1);
  int *dev_cell, *dev_field;
  int *field = world.getField();
  dev_field = transfer_data_from_host_to_gpu(field, agent_size);
  dev_cell = transfer_data_from_host_to_gpu(&(this->condition[0]), agent_size);
  game<<<grids,blocks>>>(dev_cell, dev_field, minX, maxX, minY, maxY, agent_size);
  synch_field<<<grids, blocks>>>(dev_cell, dev_field, agent_size);//field synch
  transfer_data_from_gpu_to_host(dev_cell, &(this->condition[0]), agent_size);
  transfer_data_from_gpu_to_host(dev_field, field, agent_size);
}

void Cell::print_result(){
  for (int y=minY; y<=maxY; ++y){
    if(y==minY){
      for (int i=minX; i<=(maxX+2);++i){
	printf("- ");
      }
      printf("\n");
    }   
    for (int x=minX; x<=maxX; ++x){
      if (x==minX){
	printf("| ");
      }
      if (this->condition[x+(maxX-minX+1)*y]==1){
	printf("* ");
      }
      else if (this->condition[x+(maxX-minX+1)*y]==0){
	printf("  ");
      }
      else{
	printf("cell error!\n");
	exit(1);
      }
      if (x==maxX){
	printf("|\n");
      }
    }
    if(y==maxY){
      for (int i=minX; i<=(maxX+2);++i){
	printf("- ");
      }
      printf("\n");
    }
  }
}


__global__ void synch_field(int *dev_condition, int *dev_field, int size){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < size){
    dev_field[tid] = dev_condition[tid];
  }
}


__device__ int around_me(int *world,int tid,int minX,int maxX,int minY,int maxY){
  return check_with_distance8(world,tid,minX,maxX,minY,maxY);
}


/*
ライフゲームのルール実装
*/
__global__ void game(int *dev_cell, int *dev_world, int minX, int maxX, int minY, int maxY, int N){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (tid < N){
    int around_life = around_me(dev_world, tid, minX, maxX, minY, maxY);
    
    if (dev_cell[tid] == 1){
      if (around_life <= 1){
	dev_cell[tid] = 0;
      }
      else if ((2 <= around_life) && (around_life <= 3)){
	dev_cell[tid] = 1;
      }
      else if (4 <= around_life){
	dev_cell[tid] = 0;
      }
    }
    else if (dev_cell[tid] == 0){
      if ((1 <= around_life) && (around_life <= 2)){
	dev_cell[tid] = 0;
      }
      else if (around_life == 3){
	dev_cell[tid] = 1;
      }
      else if (4 <= around_life){
	dev_cell[tid] = 0;
      }
    }
    else{
      printf("This case can't cause in tid%d:%d!", tid, dev_cell[tid]);
    }
  }
}
