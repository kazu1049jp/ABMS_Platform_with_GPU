#include <stdio.h>
#include <iostream>
#include "../../world/worldCreator.cuh"
#include "cell.cuh"
#include <cutil.h>

using namespace std;

void run(){
  worldCreator<int> world;
  world.make_2D_World();
  Cell agent(world);
  int agent_size = (world.getMax_X()-world.getMin_X()+1)*(world.getMax_Y()-world.getMin_Y()+1);

  printf("initial condition\n");
  agent.print_result();

  int step_count = 100;
  for (int i=0; i<step_count; ++i){
    printf("step%d\n",i+1);
    agent.step(agent_size, world);
    agent.print_result();
  }
  

}

int main(void){
  CUDA_SAFE_CALL(hipSetDevice(0));
  float elapsed_time_ms = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  run();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  printf("Run time from start to end : %f [sec]\n", elapsed_time_ms/1000);

  return 0;
}
