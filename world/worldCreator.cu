#include "hip/hip_runtime.h"
#include "worldCreator.cuh"
#include "worldDefinition.h"
#include <cstdlib>
#include <iostream>
#include <cutil.h>

using namespace std;


/*
  Initialize variables
*/
template <class T>
worldCreator<T>::worldCreator(){
  this->field = NULL;
  worldDefinition<T> world;
  this->min_x = world.getMin_X();
  this->max_x = world.getMax_X();
  this->min_y = world.getMin_Y();
  this->max_y = world.getMax_Y();
  T x_range = this->max_x - this->min_x + 1;
  T y_range = this->max_y - this->min_y + 1;
  this->N = x_range * y_range;
  this->field = (int*) malloc(sizeof(T)*(this->N));//デストラクタで解放
  if (this->field == NULL){
    cout << "Simulator can't get a pointer in worldCreator." << endl;
  }
}
template worldCreator<int>::worldCreator();
template worldCreator<float>::worldCreator();
template worldCreator<double>::worldCreator(); 


/*
  Free the malloced pointer
*/
template <class T>
worldCreator<T>::~worldCreator(){
  if (field != NULL){
    free(field);
  }
}
template worldCreator<int>::~worldCreator();
template worldCreator<float>::~worldCreator();
template worldCreator<double>::~worldCreator();


/*
  Set the range defined in worldDefinition
*/
template <class T>
void worldCreator<T>::setRange(T minX, T maxX, T minY, T maxY){
  this->min_x = minX;
  this->max_x = maxX;
  this->min_y = minY;
  this->max_y = maxY;
}
template void worldCreator<int>::setRange(int minX, int maxX, int minY, int maxY);
template void worldCreator<float>::setRange(float minX, float maxX, float minY, float maxY);
template void worldCreator<double>::setRange(double minX, double maxX, double minY, double maxY);

template <class T>
__global__ void fieldSet(T* field, int N){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid < N){
    field[tid] = 0;
  }
}
template __global__ void fieldSet<int>(int* field, int N);
template __global__ void fieldSet<float>(float* field, int N);
template __global__ void fieldSet<double>(double* field, int N);

/*
  Make field with the range!
  We must set detail information at the other place.
*/
template <class T>
void worldCreator<T>::make_2D_World(){
  T* dev_field;
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_field, sizeof(T)*N));
  dim3 blocks(1024, 1 ,1);
  dim3 grids((N + 1023)/1024, 1, 1);
  fieldSet<<<grids, blocks>>>(dev_field, N);
  CUDA_SAFE_CALL(hipMemcpy(field, dev_field, sizeof(T)*N, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(dev_field));
}
template void worldCreator<int>::make_2D_World();
template void worldCreator<float>::make_2D_World();
template void worldCreator<double>::make_2D_World();

template <class T>
T worldCreator<T>::getMax_X(){
  return this->max_x;
}
template int worldCreator<int>::getMax_X();
template float worldCreator<float>::getMax_X();
template double worldCreator<double>::getMax_X();

template <class T>
T worldCreator<T>::getMin_X(){
  return this->min_x;
}
template int worldCreator<int>::getMin_X();
template float worldCreator<float>::getMin_X();
template double worldCreator<double>::getMin_X();

template <class T>
T worldCreator<T>::getMax_Y(){
  return this->max_y;
}
template int worldCreator<int>::getMax_Y();
template float worldCreator<float>::getMax_Y();
template double worldCreator<double>::getMax_Y();

template <class T>
T worldCreator<T>::getMin_Y(){
  return this->min_y;
}
template int worldCreator<int>::getMin_Y();
template float worldCreator<float>::getMin_Y();
template double worldCreator<double>::getMin_Y();

template <class T>
int* worldCreator<T>::getField(){
  return &(this->field[0]);
}
template int* worldCreator<int>::getField();
template int* worldCreator<float>::getField();
template int* worldCreator<double>::getField();

template <class T>
void worldCreator<T>::setField(int* dataP, int N){
  this->field = (int*)malloc(sizeof(int)*N);
  for (int i=0; i<N; ++i){
    this->field[i] = dataP[i];
  }
}
template void worldCreator<int>::setField(int* dataP, int N);
template void worldCreator<float>::setField(int* dataP, int N);
template void worldCreator<double>::setField(int* dataP, int N);
